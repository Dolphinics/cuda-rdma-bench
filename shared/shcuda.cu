#include <hip/hip_runtime.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "shared_cuda.h"

int GetGpuCount()
{
    hipError_t status;

    int device_count = 0;

    status = hipGetDeviceCount(&device_count);

    if (hipSuccess != status)
    {
        return -1;
    }

    return device_count;
}



void* GetGpuDevicePtr(const void* ptr)
{
    hipPointerAttribute_t attrs;

    hipError_t err = hipPointerGetAttributes(&attrs, ptr);

    if (hipSuccess != err)
    {
        return NULL;
    }

    return attrs.devicePointer;
}



int SetSyncMemops(void* ptr)
{
    uint32_t flag = 1;
    hipError_t res = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) ptr);

    if (hipSuccess != res)
    {
        return -1;
    }

    return 0;
}
