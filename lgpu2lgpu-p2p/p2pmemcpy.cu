#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>
#include "shared_cuda.h"

static size_t factor = 1000L;

void EnableP2P(int dev, int peer)
{
    hipSetDevice(dev);
    
    int access = 0;
    hipDeviceCanAccessPeer(&access, dev, peer);
    cudaCheckError();

    if (access)
    {
        hipDeviceEnablePeerAccess(peer, 0);
        cudaCheckError();
    }
}

void DisableP2P(int dev, int peer)
{
    hipSetDevice(dev);

    int access = 0;
    hipDeviceCanAccessPeer(&access, dev, peer);
    cudaCheckError();

    if (access)
    {
        hipDeviceDisablePeerAccess(peer);
        cudaCheckError();
    }
}

void ConfigureP2P(int ctl, int src, int dst, int bidirect, int useP2P)
{
    int dev1, dev2;

    if (ctl == src)
    {
        dev1 = src;
        dev2 = dst;
    }
    else
    {
        dev1 = dst;
        dev2 = src;
    }

    if (useP2P)
    {
        EnableP2P(dev1, dev2);
        if (bidirect)
        {
            EnableP2P(dev2, dev1);
        }
    }
    else
    {
        //DisableP2P(dev1, dev2);
        //if (bidirect)
        /*{
            DisableP2P(dev2, dev1);
        }*/
    }
}


void AllocHostAndDevBufs(int dev, uint8_t** devptr, uint8_t** hostptr, size_t size, int pinned, unsigned type, hipStream_t* stream)
{
    hipSetDevice(dev);
    cudaCheckError();

    if (pinned)
    {
        hipHostAlloc((void**) hostptr, size, type);
        cudaCheckError();
    }
    else
    {
        *hostptr = (uint8_t*) malloc(size);
        if (*hostptr == NULL)
        {
            perror("malloc");
            exit(1);
        }
    }

    if (pinned && !!(type & hipHostMallocMapped))
    {
        hipHostGetDevicePointer((void**) devptr, (void*) *hostptr, 0);
        cudaCheckError();
    }
    else
    {
        hipMalloc((void**) devptr, size);
        cudaCheckError();
    }

    hipStreamCreate(stream);
    cudaCheckError();
}

void PrepareBenchmark(int dev, hipEvent_t* start, hipEvent_t* stop)
{
    hipSetDevice(dev);

    hipEventCreate(start);
    cudaCheckError();
    hipEventCreate(stop);
    cudaCheckError();

    hipDeviceSynchronize();
    cudaCheckError();
}

double GigabytesPerSecond(hipEvent_t start, hipEvent_t stop, size_t size, int repeats)
{
    float time_ms;
    double time_s, gigabytes;
    
    hipEventElapsedTime(&time_ms, start, stop);
    time_s = time_ms / (double) 1e3;
    gigabytes = (size * repeats) / (double) (factor * factor * factor);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return gigabytes / time_s;
}

double Microseconds(hipEvent_t start, hipEvent_t stop, int repeats)
{
    float time_ms;
    double time_us;

    hipEventElapsedTime(&time_ms, start, stop);
    time_us = (time_ms * 1e3) / ((double) repeats);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return time_us;
}

void MeasureLatency(int ctlDev, int srcDev, int dstDev, int pinned, unsigned memType, int repeat)
{
    double usecs;
    hipEvent_t start, stop;

    uint8_t *srcBuf, *srcPtr;
    hipStream_t srcStream;
    AllocHostAndDevBufs(srcDev, &srcPtr, &srcBuf, 1, pinned, memType, &srcStream);

    uint8_t *dstBuf, *dstPtr;
    hipStream_t dstStream;
    AllocHostAndDevBufs(dstDev, &dstPtr, &dstBuf, 1, pinned, memType, &dstStream);

    // HOST TO DEVICE
    PrepareBenchmark(srcDev, &start, &stop);

    hipEventRecord(start);
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyAsync(srcPtr, srcBuf, 1, hipMemcpyHostToDevice);
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    usecs = Microseconds(start, stop, repeat);
    printf("Host to device  : %6.02f %s\n", usecs, "µs");

    
    // DEVICE TO DEVICE
    PrepareBenchmark(ctlDev, &start, &stop);

    hipEventRecord(start);
    // hipMemcpyPeerAsync will fall back to hipMemcpyAsync when p2p is disabled
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyPeerAsync(dstPtr, dstDev, srcPtr, srcDev, 1, dstStream);
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    usecs = Microseconds(start, stop, repeat);
    printf("Device to device: %6.02f %s\n", usecs, "µs");


    // DEVICE TO HOST
    PrepareBenchmark(dstDev, &start, &stop);

    hipEventRecord(start);
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyAsync(dstBuf, dstPtr, 1, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    usecs = Microseconds(start, stop, repeat);
    printf("Device to host  : %6.02f %s\n", usecs, "µs");


    hipHostFree(srcBuf);
    hipHostFree(dstBuf);

    hipSetDevice(srcDev);
    hipFree(srcPtr);
    hipStreamDestroy(srcStream);

    hipSetDevice(dstDev);
    hipFree(dstPtr);
    hipStreamDestroy(dstStream);
}

void MeasureBandwidth(int ctlDev, int srcDev, int dstDev, size_t memSize, int bidirectional, int pinned, unsigned memType, int repeat, int verify)
{
    double gbps;
    hipEvent_t start, stop;

    uint8_t *srcBuf, *srcPtr;
    hipStream_t srcStream;
    AllocHostAndDevBufs(srcDev, &srcPtr, &srcBuf, memSize, pinned, memType, &srcStream);

    uint8_t *dstBuf, *dstPtr;
    hipStream_t dstStream;
    AllocHostAndDevBufs(dstDev, &dstPtr, &dstBuf, memSize, pinned, memType, &dstStream);

    if (verify)
    {
        for (size_t i = 0; i < memSize; ++i)
        {
            srcBuf[i] = rand() & 255;
        }
    }

    // Host to device
    PrepareBenchmark(srcDev, &start, &stop);

    hipEventRecord(start);
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyAsync(srcPtr, srcBuf, memSize, hipMemcpyHostToDevice);
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    gbps = GigabytesPerSecond(start, stop, memSize, repeat);
    printf("Host to device  : %6.02f %s\n", gbps, factor == 1024L ? "GiB/s" : "GB/s");


    // Device to device
    PrepareBenchmark(ctlDev, &start, &stop);

    hipEventRecord(start);
    // hipMemcpyPeerAsync will fall back to hipMemcpyAsync when p2p is disabled
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyPeerAsync(dstPtr, dstDev, srcPtr, srcDev, memSize, dstStream);
        if (bidirectional)
        {
            hipMemcpyPeerAsync(srcPtr, srcDev, dstPtr, dstDev, memSize, srcStream);
        }
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    gbps = GigabytesPerSecond(start, stop, memSize, repeat);
    printf("Device to device: %6.02f %s\n", gbps, factor == 1024L ? "GiB/s" : "GB/s");


    // Device to host
    PrepareBenchmark(dstDev, &start, &stop);

    hipEventRecord(start);
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyAsync(dstBuf, dstPtr, memSize, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop);

    hipDeviceSynchronize();
    cudaCheckError();

    gbps = GigabytesPerSecond(start, stop, memSize, repeat);
    printf("Device to host  : %6.02f %s\n", gbps, factor == 1024L ? "GiB/s" : "GB/s");


    if (verify)
    {
        size_t i;
        for (i = 0; i < memSize && srcBuf[i] == dstBuf[i]; ++i);

        if (i != memSize)
        {
            printf("\n ***** Data was NOT transfered properly! Byte %ld differs! *****\n", i);
        }
    }

    hipHostFree(srcBuf);
    hipHostFree(dstBuf);

    hipSetDevice(srcDev);
    hipFree(srcPtr);
    hipStreamDestroy(srcStream);

    hipSetDevice(dstDev);
    hipFree(dstPtr);
    hipStreamDestroy(dstStream);
}

void ListDevices()
{
    int count;

    hipGetDeviceCount(&count);
    cudaCheckError();

    int** peerAccessMatrix = (int**) malloc(sizeof(int*) * count);

    for (int i = 0; i < count; ++i)
    {
        hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);
        cudaCheckError();

        printf("%3d %-16s  %02x:%02x.%x\n",
            i, prop.name, prop.pciBusID, prop.pciDomainID, prop.pciDeviceID);

        peerAccessMatrix[i] = (int*) malloc(sizeof(int) * count);

        for (int j = 0; j < count; ++j)
        {
            if (i != j)
            {
                hipSetDevice(i);
                cudaCheckError();

                hipDeviceCanAccessPeer(&peerAccessMatrix[i][j], j, 0);
                cudaCheckError();
            }
        }
    }

    printf("\nP2P");
    for (int i = 0; i < count; ++i)
    {
        printf(" %3d", i);
    }
    printf("\n");

    for (int i = 0; i < count; ++i)
    {
        printf("%3d", i);
        for (int j = 0; j < count; ++j)
        {
            if (i == j)
            {
                printf("   -");
            }
            else if (peerAccessMatrix[i][j])
            {
                printf("   y");
            }
            else
            {
                printf("   n");
            }
        }
        printf("\n");

        free(peerAccessMatrix[i]);
    }
    free(peerAccessMatrix);
}

int main(int argc, char** argv)
{
    // Parameters
    size_t size = 0;
    int srcDevice = -1;
    int dstDevice = -1;
    int bidirectional = 0;
    int usePeer2Peer = 0;
    int oppositeDevice = 0;
    int verify = 0;
    int repeat = 5;
    unsigned memtype = hipHostMallocDefault;
    int pinned = 1;

    // Get device count
    int devCount = 0;
    hipError_t err;
    err = hipGetDeviceCount(&devCount);
    switch (err)
    {
        case hipErrorNoDevice:
            fprintf(stderr, "No CUDA capable device detected!\n");
            return 1;

        default:
            cudaCheckError();
            break;
    }

    // Parse command line options
    struct option opts[] = {
        { .name = "srcdev", .has_arg = 1, .flag = NULL, .val = 1 },
        { .name = "dstdev", .has_arg = 1, .flag = NULL, .val = 2 },
        { .name = "size", .has_arg = 1, .flag = NULL, .val = 3 },
        { .name = "peer", .has_arg = 0, .flag = NULL, .val = 4 },
        { .name = "help", .has_arg = 0, .flag = NULL, .val = 'h' },
    };
    int opt, optidx;
    char* strptr;

    while ((opt = getopt_long(argc, argv, "-:pmwsvbor:ilh", opts, &optidx)) != -1)
    {
        switch (opt)
        {
            case ':': // missing value
                fprintf(stderr, "Argument %s requires an argument\n", argv[optind - 1]);
                goto giveUsage;

            case '?': // unknown flag
                fprintf(stderr, "Unknown option: -%c\n", optopt);
                goto giveUsage;

            case 1: // set source device
                strptr = NULL;
                srcDevice = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || srcDevice >= devCount)
                {
                    fprintf(stderr, "Argument --srcdev requires a valid CUDA device number\n");
                    goto giveUsage;
                }
                else if (dstDevice == srcDevice)
                {
                    fprintf(stderr, "NOTE!! Source device is equal to destination device!\n");
                }
                break;

            case 2: // set destination device
                strptr = NULL;
                dstDevice = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || dstDevice >= devCount)
                {
                    fprintf(stderr, "Argument --dstdev requires a valid CUDA device number\n");
                    goto giveUsage;
                }
                else if (dstDevice == srcDevice)
                {
                    fprintf(stderr, "NOTE!! Destination device is equal to source device!\n");
                }
                break;

            case 3: // set memory chunk size 
                strptr = NULL;
                size = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || size == 0)
                {
                    fprintf(stderr, "Argument --size requires a valid memory size\n");
                    return 1;
                }
                break;

            case 4: // enable peer-to-peer
                usePeer2Peer = 1;
                break;

            case 'p':
                memtype |= hipHostMallocPortable;
                break;

            case 'm':
                memtype |= hipHostMallocMapped;
                break;

            case 'w':
                memtype |= hipHostMallocWriteCombined;
                break;

            case 's':
                pinned = 0;
                break;

            case 'v': // verify transfer 
                verify = 1;
                break;

            case 'b': // bidirectional benchmark (if p2p is supported+enabled, this will show the difference)
                bidirectional = 1;
                break;

            case 'o': // reverse hipSetDevice
                oppositeDevice = 1;
                break;

            case 'r': // set number of times to repeat
                strptr = NULL;
                repeat = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || repeat <= 0)
                {
                    fprintf(stderr, "Option -r requires a valid number equal to or larger than 1\n");
                    return 1;
                }
                break;

            case 'i': // use MiBs instead of MBs
                factor = 1024L;
                break;

            case 'l': // list CUDA enabled devices
                ListDevices();
                return 0;

            case 'h': // show help
                goto giveUsage;
        }
    }

    // Verify program arguments
    if (srcDevice < 0 || dstDevice < 0 || size == 0)
    {
        fprintf(stderr, "Missing required arguments!!\n");
        goto giveUsage;
    }

    // Calculate chunk size
    size = size * factor * factor;

    if (!!(memtype & hipHostMallocMapped))
    {
        hipSetDevice(srcDevice);
        hipSetDeviceFlags(hipDeviceMapHost);
        cudaCheckError();
        hipSetDevice(dstDevice);
        hipSetDeviceFlags(hipDeviceMapHost);
        cudaCheckError();
    }
    
    ConfigureP2P(
            oppositeDevice ? dstDevice : srcDevice, 
            srcDevice,
            dstDevice,
            bidirectional,
            usePeer2Peer
            );

    MeasureBandwidth(
            oppositeDevice ? dstDevice : srcDevice, 
            srcDevice, 
            dstDevice, 
            size, 
            bidirectional, 
            pinned,
            memtype,
            repeat,
            verify
            );

    MeasureLatency(
            oppositeDevice ? dstDevice : srcDevice, 
            srcDevice, 
            dstDevice, 
            pinned,
            memtype,
            repeat * 1e4 
            );

    return 0;

giveUsage:
    fprintf(stderr, 
            "Usage: %s --srcdev=<device no> --dstdev=<device no> --size=<size> [--peer] [-pmw|-s] [-vboil] [-r <number>]\n"
            "\nArguments\n"
            "  --srcdev=<device no>  CUDA device to copy data from\n"
            "  --dstdev=<device no>  CUDA device to copy data to\n"
            "  --size=<size>         memory chunk size in MB (or MiB if -i is set)\n"
            "  --peer                enable p2p if possible\n"
            "\nOptions\n"
            "   -p                   use hipHostMallocPortable flag\n"
            "   -m                   use hipHostMallocMapped flag\n"
            "   -w                   use hipHostMallocWriteCombined flag\n"
            "   -s                   use pagable system memory (malloc)\n"
            "   -v                   verify transfer by copying memory from device and comparing\n"
            "   -b                   transfer memory in both directions simultaneously\n"
            "   -o                   make hipSetDevice set opposite device (pull instead of push)\n"
            "   -r <number>          number of times to repeat (default is 5)\n"
            "   -i                   use IEC units (1024) instead of SI units (1000)\n"
            "   -l                   list CUDA devices and quit\n"
            "\nBuild date: %s %s\n"
            , argv[0], __DATE__, __TIME__);

    return 1;
}
