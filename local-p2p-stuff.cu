#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#define cudaCheckError()                                \
    do {                                                \
        hipError_t __err__ = hipGetLastError();       \
        if (__err__ != hipSuccess) {                   \
            fprintf(stderr, "CUDA error %s:%d: %s\n",  \
                __FILE__, __LINE__,                     \
                hipGetErrorString(__err__));           \
            exit(1);                                    \
        }                                               \
    } while (0)

#define STRINGIFY(s) #s

static size_t factor = 1000L;

void EnableP2P(int dev, int peer)
{
    hipSetDevice(dev);
    
    int access = 0;
    hipDeviceCanAccessPeer(&access, dev, peer);
    cudaCheckError();

    if (access)
    {
        hipDeviceEnablePeerAccess(peer, 0);
        cudaCheckError();
    }
}

void DisableP2P(int dev, int peer)
{
    hipSetDevice(dev);

    int access = 0;
    hipDeviceCanAccessPeer(&access, dev, peer);
    cudaCheckError();

    if (access)
    {
        hipDeviceDisablePeerAccess(peer);
        cudaCheckError();
    }
}

void ConfigureP2P(int ctl, int src, int dst, int bidirect, int useP2P)
{
    int dev1, dev2;

    if (ctl == src)
    {
        dev1 = src;
        dev2 = dst;
    }
    else
    {
        dev1 = dst;
        dev2 = src;
    }

    if (useP2P)
    {
        EnableP2P(dev1, dev2);
        if (bidirect)
        {
            EnableP2P(dev2, dev1);
        }
    }
    else
    {
        DisableP2P(dev1, dev2);
        if (bidirect)
        {
            DisableP2P(dev2, dev1);
        }
    }
}


void AllocHostAndDevBufs(int dev, uint8_t** devptr, uint8_t** hostptr, size_t size, unsigned type, hipStream_t* stream)
{
    hipSetDevice(dev);
    cudaCheckError();

    hipHostAlloc((void**) hostptr, size, type);
    cudaCheckError();

    if (!!(type & hipHostMallocMapped))
    {
        hipHostGetDevicePointer((void**) devptr, (void*) *hostptr, 0);
        cudaCheckError();
    }
    else
    {
        hipMalloc((void**) devptr, size);
        cudaCheckError();
    }

    hipStreamCreate(stream);
    cudaCheckError();
}

void MeasureBandwidth(int ctlDev, int srcDev, int dstDev, size_t memSize, int bidirectional, int p2p, unsigned memType, int repeat, int verify)
{
    float time_ms;
    double time_s, gigabytes;

    uint8_t *srcBuf, *srcPtr;
    hipStream_t srcStream;
    AllocHostAndDevBufs(srcDev, &srcPtr, &srcBuf, memSize, memType, &srcStream);

    uint8_t *dstBuf, *dstPtr;
    hipStream_t dstStream;
    AllocHostAndDevBufs(dstDev, &dstPtr, &dstBuf, memSize, memType, &dstStream);

    ConfigureP2P(ctlDev, srcDev, dstDev, bidirectional, p2p);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    cudaCheckError();
    hipEventCreate(&stop);
    cudaCheckError();

    if (verify)
    {
        for (size_t i = 0; i < memSize; ++i)
        {
            srcBuf[i] = rand() & 255;
        }

        if (!(memType & hipHostMallocMapped))
        {
            hipSetDevice(srcDev);
            hipDeviceSynchronize();
            cudaCheckError();

            hipEventRecord(start);
            for (int i = 0; i < repeat; ++i)
            {
                hipMemcpyAsync(srcPtr, srcBuf, memSize, hipMemcpyHostToDevice);
            }
            hipEventRecord(stop);

            hipDeviceSynchronize();
            cudaCheckError();

            hipEventElapsedTime(&time_ms, start, stop);
            time_s = time_ms / (double) 1e3;
            gigabytes = (memSize * repeat) / (double) (factor * factor * factor);

            printf("Host to device  : %6.02f %s\n", gigabytes / time_s, factor == 1024L ? "GiB/s" : "GB/s");
        }
        else
        {
            printf("Host to device  : memory is mapped\n");
        }
    }

    hipSetDevice(ctlDev);
    hipDeviceSynchronize();
    cudaCheckError();
    hipEventRecord(start);

    // cudaMemcpyPeerAsync will fall back to cudaMemcpyAsync when p2p is disabled
    for (int i = 0; i < repeat; ++i)
    {
        hipMemcpyPeerAsync(dstPtr, dstDev, srcPtr, srcDev, memSize, dstStream);
        if (bidirectional)
        {
            hipMemcpyPeerAsync(srcPtr, srcDev, dstPtr, dstDev, memSize, srcStream);
        }
    }

    hipEventRecord(stop);
    hipDeviceSynchronize();
    cudaCheckError();

    hipEventElapsedTime(&time_ms, start, stop);
    time_s = time_ms / (double) 1e3;
    gigabytes = (memSize * repeat) / (double) (factor * factor * factor);

    printf("Device to device: %6.02f %s\n", gigabytes / time_s, factor == 1024L ? "GiB/s" : "GB/s");

    if (verify)
    {
        if (!(memType & hipHostMallocMapped))
        {
            hipSetDevice(srcDev);
            hipDeviceSynchronize();
            cudaCheckError();

            hipEventRecord(start);
            for (int i = 0; i < repeat; ++i)
            {
                hipMemcpyAsync(dstBuf, srcPtr, memSize, hipMemcpyDeviceToHost);
            }
            hipEventRecord(stop);

            hipDeviceSynchronize();
            cudaCheckError();

            hipEventElapsedTime(&time_ms, start, stop);
            time_s = time_ms / (double) 1e3;
            gigabytes = (memSize * repeat) / (double) (factor * factor * factor);

            printf("Device to host  : %6.02f %s\n", gigabytes / time_s, factor == 1024L ? "GiB/s" : "GB/s");
        }
        else
        {
            printf("Device to host  : memory is mapped\n");
        }

        size_t i;
        for (i = 0; i < memSize && srcBuf[i] == dstBuf[i]; ++i);

        if (i != memSize)
        {
            printf("\n ***** Data was NOT transfered properly! Byte %ld differs! *****\n", i);
        }
    }

    hipHostFree(srcBuf);
    hipHostFree(dstBuf);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipSetDevice(srcDev);
    hipFree(srcPtr);
    hipStreamDestroy(srcStream);

    hipSetDevice(dstDev);
    hipFree(dstPtr);
    hipStreamDestroy(dstStream);
}

void ListDevices()
{
    int count;

    hipGetDeviceCount(&count);
    cudaCheckError();

    int** peerAccessMatrix = (int**) malloc(sizeof(int*) * count);

    for (int i = 0; i < count; ++i)
    {
        hipDeviceProp_t prop;

        hipGetDeviceProperties(&prop, i);
        cudaCheckError();

        printf("%3d %-16s  %02x:%02x.%x\n",
            i, prop.name, prop.pciBusID, prop.pciDomainID, prop.pciDeviceID);

        peerAccessMatrix[i] = (int*) malloc(sizeof(int) * count);

        for (int j = 0; j < count; ++j)
        {
            if (i != j)
            {
                hipSetDevice(i);
                cudaCheckError();

                hipDeviceCanAccessPeer(&peerAccessMatrix[i][j], j, 0);
                cudaCheckError();
            }
        }
    }

    printf("\nP2P");
    for (int i = 0; i < count; ++i)
    {
        printf(" %3d", i);
    }
    printf("\n");

    for (int i = 0; i < count; ++i)
    {
        printf("%3d", i);
        for (int j = 0; j < count; ++j)
        {
            if (i == j)
            {
                printf("   -");
            }
            else if (peerAccessMatrix[i][j])
            {
                printf("   y");
            }
            else
            {
                printf("   n");
            }
        }
        printf("\n");

        free(peerAccessMatrix[i]);
    }
    free(peerAccessMatrix);
}

int main(int argc, char** argv)
{
    // Parameters
    size_t size = 0;
    int srcDevice = -1;
    int dstDevice = -1;
    int bidirectional = 0;
    int usePeer2Peer = 0;
    int oppositeDevice = 0;
    int verify = 0;
    int repeat = 5;
    unsigned memtype = hipHostMallocDefault;

    // Get device count
    int devCount = 0;
    hipError_t err;
    err = hipGetDeviceCount(&devCount);
    switch (err)
    {
        case hipErrorNoDevice:
            fprintf(stderr, "No CUDA capable device detected!\n");
            return 1;

        default:
            cudaCheckError();
            break;
    }

    // Parse command line options
    struct option opts[] = {
        { .name = "srcdev", .has_arg = 1, .flag = NULL, 1 },
        { .name = "dstdev", .has_arg = 1, .flag = NULL, 2 },
        { .name = "size", .has_arg = 1, .flag = NULL, 3 },
        { .name = "peer", .has_arg = 0, .flag = NULL, 4 },
        { .name = "help", .has_arg = 0, .flag = NULL, 'h' },
    };
    int opt, optidx;
    char* strptr;

    while ((opt = getopt_long(argc, argv, "-:hbpiolvmwkr:", opts, &optidx)) != -1)
    {
        switch (opt)
        {
            case ':': // missing value
                fprintf(stderr, "Argument %s requires an argument\n", argv[optind - 1]);
                goto giveUsage;

            case '?': // unknown flag
                fprintf(stderr, "Unknown option: -%c\n", optopt);
                goto giveUsage;

            case 1: // set source device
                strptr = NULL;
                srcDevice = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || srcDevice >= devCount)
                {
                    fprintf(stderr, "Argument --srcdev requires a valid CUDA device number\n");
                    goto giveUsage;
                }
                else if (dstDevice == srcDevice)
                {
                    fprintf(stderr, "NOTE!! Source device is equal to destination device!\n");
                }
                break;

            case 2: // set destination device
                strptr = NULL;
                dstDevice = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || dstDevice >= devCount)
                {
                    fprintf(stderr, "Argument --dstdev requires a valid CUDA device number\n");
                    goto giveUsage;
                }
                else if (dstDevice == srcDevice)
                {
                    fprintf(stderr, "NOTE!! Destination device is equal to source device!\n");
                }
                break;

            case 3: // set memory chunk size 
                strptr = NULL;
                size = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || size == 0)
                {
                    fprintf(stderr, "Argument --size requires a valid memory size\n");
                    return 1;
                }
                break;

            case 4: // enable peer-to-peer
                usePeer2Peer = 1;
                break;

            case 'v': // verify transfer 
                verify = 1;
                break;

            case 'b': // bidirectional benchmark (if p2p is supported+enabled, this will show the difference)
                bidirectional = 1;
                break;

            case 'p':
                memtype |= hipHostMallocPortable;
                break;

            case 'm':
                memtype |= hipHostMallocMapped;
                break;

            case 'w':
                memtype |= hipHostMallocWriteCombined;
                break;

            case 'o': // reverse cudaSetDevice
                oppositeDevice = 1;
                break;

            case 'r': // set number of times to repeat
                strptr = NULL;
                repeat = strtoul(optarg, &strptr, 0);
                if (strptr == NULL || *strptr != '\0' || repeat <= 0 || repeat > 1000)
                {
                    fprintf(stderr, "Option -r requires a valid number between 1 and 1000\n");
                    return 1;
                }
                break;

            case 'i': // use MiBs instead of MBs
                factor = 1024L;
                break;

            case 'l': // list CUDA enabled devices
                ListDevices();
                return 0;

            case 'h': // show help
                goto giveUsage;
        }
    }

    // Verify program arguments
    if (srcDevice < 0 || dstDevice < 0 || size == 0)
    {
        fprintf(stderr, "Missing required arguments!!\n");
        goto giveUsage;
    }

    // Calculate chunk size
    size = size * factor * factor;

    // If 
    if (!!(memtype & hipHostMallocMapped))
    {
        hipSetDevice(srcDevice);
        hipSetDeviceFlags(hipDeviceMapHost);
        cudaCheckError();
        hipSetDevice(dstDevice);
        hipSetDeviceFlags(hipDeviceMapHost);
        cudaCheckError();
    }
    
    // Allocate host buffer
    MeasureBandwidth(
            oppositeDevice ? dstDevice : srcDevice, 
            srcDevice, 
            dstDevice, 
            size, 
            bidirectional, 
            usePeer2Peer,
            memtype,
            repeat,
            verify
    );

    return 0;

giveUsage:
    fprintf(stderr, 
            "Usage: %s --srcdev=<device no> --dstdev=<device no> --size=<size> [--peer] [options]\n"
            "\nArguments\n"
            "  --srcdev=<device no>  CUDA device to copy data from\n"
            "  --dstdev=<device no>  CUDA device to copy data to\n"
            "  --size=<size>         memory chunk size in MB (or MiB if -i is set)\n"
            "  --peer                enable p2p is possible\n"
            "\nOptions\n"
            "   -v                   verify transfer by copying memory from device and comparing\n"
            "   -b                   transfer memory in both directions simultaneously\n"
            "   -p                   use hipHostMallocPortable flag\n"
            "   -m                   use hipHostMallocMapped flag\n"
            "   -w                   use hipHostMallocWriteCombined flag\n"
            "   -o                   make hipSetDevice set opposite device (pull instead of push)\n"
            "   -r <number>          number of times to repeat (default is 5)\n"
            "   -i                   use IEC units (1024) instead of SI units (1000)\n"
            "   -l                   list CUDA devices and quit\n"
            "\nBuild date: %s %s\n"
            , argv[0], __DATE__, __TIME__);

    return 1;
}