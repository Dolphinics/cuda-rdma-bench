#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sisci_api.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "gpu.h"
#include "reporting.h"


__global__ void gpu_memset_kernel(void* buf, size_t len, uint8_t val)
{
    const int num = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int pos = y * (gridDim.x * blockDim.x) + x;

    uint8_t* ptr = (uint8_t*) buf;

    for (size_t i = pos * (len / num), n = (pos + 1) * (len / num); i < n && i < len; ++i)
    {
        ptr[i] = val;
    }

    __syncthreads();
}


__global__ void gpu_memcmp_kernel(void* local, void* remote, size_t len, uint8_t* result)
{
    int num = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pos = y * (gridDim.x * blockDim.x) + x;

    uint8_t* l_ptr = (uint8_t*) local;
    uint8_t* r_ptr = (uint8_t*) remote;

    size_t i = pos * (len / num);
    size_t n = (pos + 1) * (len / num);

    for ( ; i < n && i < len && l_ptr[i] == r_ptr[i]; ++i);

    __syncthreads();

    result[pos] = i == n;
}


extern "C"
void gpu_memset(int gpu, void* ptr, size_t len, uint8_t val)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    dim3 grid;
    grid.x = 4;
    grid.y = 4;

    dim3 block;
    block.x = 4;
    block.y = 4;

    log_debug("Filling buffer with value %02x...", val);

    gpu_memset_kernel<<<grid, block>>>(ptr, len, val);

    hipDeviceSynchronize();
}


// Copy remote buffer to GPU buffer and do memcmp in parallel
int gpu_memcmp_gpu(int gpu, void* local, void* remote, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    dim3 grid;
    grid.x = 4;
    grid.y = 4;

    dim3 block;
    block.x = 4;
    block.y = 4;

    uint8_t* result = NULL;
    err = hipHostAlloc(&result, 32, hipHostMallocMapped);
    if (err != hipSuccess)
    {
        log_error("Out of resources: %s", hipGetErrorString(err));
        hipHostFree(result);
        return 0;
    }

    size_t i = 0, n = 256;
    for ( ; i < n; ++i)
    {
        result[i] = 0;
    }

    log_debug("Comparing local GPU memory %p to GPU-copied memory %p", local, remote);
    gpu_memcmp_kernel<<<grid, block>>>(local, remote, len, result);

    hipDeviceSynchronize();

    for (i = 0; i < n && result[i] != 0; ++i);

    hipHostFree(result);
    return i != n;
}


// Copy GPU-bound buffer to RAM and do regular memcmp
int gpu_memcmp_ram(int gpu, void* gpuptr, volatile void* ramptr, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    uint8_t* buf = NULL;
    err = hipHostAlloc(&buf, len, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate host memory: %s", hipGetErrorString(err));
        return 0;
    }

    err = hipMemcpy(buf, gpuptr, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    log_debug("Comparing local GPU memory %p to remote memory %p", gpuptr, ramptr);
    int equality = memcmp(buf, (void*) ramptr, len);

    hipHostFree(buf);
    return equality;
}


extern "C"
int gpu_memcmp(int gpu, void* local, volatile void* remote, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    void* buf = NULL;
    err = hipMalloc(&buf, len);

    if (err != hipSuccess)
    {
        log_debug("Failed to allocate buffer on device, falling back on memcmp");
        return gpu_memcmp_ram(gpu, local, remote, len);
    }

    log_debug("Copying remote memory to local GPU buffer...");
    err = hipMemcpy(buf, (void*) remote, len, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        log_error("Failed to copy from remote buffer to GPU buffer, falling back on memcmp");
        hipFree(buf);
        return gpu_memcmp_ram(gpu, local, remote, len);
    }

    int result = gpu_memcmp_gpu(gpu, local, buf, len);
    hipFree(buf);
    return result;
}


extern "C"
int gpu_device_count()
{
    hipError_t err;
    int count = 0;

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess)
    {
        log_error("Something went wrong: %s", hipGetErrorString(err));
        return -1;
    }

    return count;
}


extern "C"
void* gpu_malloc(int gpu, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }

    void* buf = NULL;
    err = hipMalloc(&buf, len);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate device memory: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("Allocated device buffer %p", buf);
    return buf;
}


extern "C"
void gpu_free(int gpu, void* ptr)
{
    log_debug("Freing buffer %p", ptr);
    hipSetDevice(gpu);
    hipFree(ptr);
}


extern "C"
void devptr_set_sync_memops(void* dev_ptr)
{
    unsigned flag = 1;
    
    hipError_t err = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) dev_ptr);

    if (err != hipSuccess)
    {
        log_error("Failed to set pointer attribute CU_POINTER_ATTRIBYTE_SYNC_MEMOPS");
    }
}


extern "C"
void* gpu_devptr(int gpu, void* ptr)
{
    hipPointerAttribute_t attrs;

    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }
        
    err = hipPointerGetAttributes(&attrs, ptr);
    if (err != hipSuccess)
    {
        log_error("Failed to get pointer attributes: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("CUDA device buffer %p has device ptr %p", ptr, attrs.devicePointer);
    return attrs.devicePointer;
}


extern "C"
void gpu_memcpy_buffer_to_local(int gpu, void* gpu_buf, void* ram_buf, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();

    err = hipMemcpy(ram_buf, gpu_buf, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();
}



extern "C"
int gpu_prepare_memcpy(int gpu, unsigned flags, volatile void* remote_ptr, size_t remote_size)
{
    hipError_t err;

    if (!!(flags & SCI_FLAG_IO_MAP_IOSPACE))
    {
        log_debug("Trying to register remote memory with CUDA driver");
        err = hipHostRegister((void*) remote_ptr, remote_size, hipHostRegisterIoMemory);
        if (err != hipSuccess)
        {
            log_warn("Failed to register remote memory with CUDA driver: %s", hipGetErrorString(err));
        }
    }
    
    err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 1;
    }

    hipDeviceSynchronize();
    return 0;
}



extern "C"
uint64_t gpu_memcpy_remote_to_local(volatile void* local_buf, volatile void* remote_buf, size_t len, int clear)
{
    hipError_t err;
    hipEvent_t before;
    hipEvent_t after;
    uint64_t elapsed = 0;
    float ms = 0;

    if (clear)
    {
        hipDeviceSynchronize();
    }

    err = hipEventCreateWithFlags(&before, hipEventDefault);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto leave;
    }

    err = hipEventCreateWithFlags(&after, hipEventBlockingSync);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_before;
    }
    
    err = hipEventRecord(before);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    err = hipMemcpy((void*) local_buf, (void*) remote_buf, len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    err = hipEventRecord(after);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    hipEventSynchronize(after);

    err = hipEventElapsedTime(&ms, before, after);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    elapsed = (uint64_t) (ms * 1e3);

destroy_after:
    hipEventDestroy(after);

destroy_before:
    hipEventDestroy(before);

leave:
    return elapsed;
}


extern "C"
uint64_t gpu_memcpy_local_to_remote(volatile void* local_buf, volatile void* remote_buf, size_t len, int clear)
{
    hipError_t err;
    hipEvent_t before;
    hipEvent_t after;
    uint64_t elapsed = 0;
    float ms = 0;

    if (clear)
    {
        hipDeviceSynchronize();
    }

    err = hipEventCreateWithFlags(&before, hipEventDefault);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto leave;
    }

    err = hipEventCreateWithFlags(&after, hipEventBlockingSync);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_before;
    }
    
    err = hipEventRecord(before);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }
    
    err = hipMemcpy((void*) remote_buf, (void*) local_buf, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    err = hipEventRecord(after);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    hipEventSynchronize(after);

    err = hipEventElapsedTime(&ms, before, after);
    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
        goto destroy_after;
    }

    elapsed = (uint64_t) (ms * 1e3);

destroy_after:
    hipEventDestroy(after);

destroy_before:
    hipEventDestroy(before);

leave:
    return elapsed;
}


extern "C"
int gpu_info(int gpu, gpu_info_t* info)
{
    hipError_t err;
    hipDeviceProp_t prop;

    err = hipGetDeviceProperties(&prop, gpu);
    if (err != hipSuccess)
    {
        log_error("Unknown GPU %d: %s", gpu, hipGetErrorString(err));
        return 0;
    }

    info->id = gpu;
    strncpy(info->name, prop.name, 256);
    info->domain = prop.pciBusID;
    info->bus = prop.pciDomainID;
    info->device = prop.pciDeviceID;

    return 1;
}
