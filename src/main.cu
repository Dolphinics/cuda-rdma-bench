#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <sisci_api.h>
#include <signal.h>
#include "translist.h"
#include "reporting.h"
#include "common.h"
#include "gpu.h"
#include "util.h"
#include "bench.h"

#define DEFAULT_LOCAL_SEGMENT_COUNT 4


/* Program options */
static struct option options[] = {
    { .name = "adapter", .has_arg = 1, .flag = NULL, .val = 'a' },
    { .name = "adapt", .has_arg = 1, .flag = NULL, .val = 'a' },
    { .name = "remote-node", .has_arg = 1, .flag = NULL, .val = 'n' },
    { .name = "remote-node-id", .has_arg = 1, .flag = NULL, .val = 'n' },
    { .name = "rni", .has_arg = 1, .flag = NULL, .val = 'n' },
    { .name = "rn", .has_arg = 1, .flag = NULL, .val = 'n' },
    { .name = "rnode", .has_arg = 1, .flag = NULL, .val = 'n' },
    { .name = "local-id", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "local-segment", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "local-segment-id", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "lsi", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "ls", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "lseg", .has_arg = 1, .flag = NULL, .val = 'l' },
    { .name = "remote-segment-id", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "remote-segment", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "remote-id", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "rsi", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "rs", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "rseg", .has_arg = 1, .flag = NULL, .val = 'r' },
    { .name = "size", .has_arg = 1, .flag = NULL, .val = 's' },
    { .name = "sz", .has_arg = 1, .flag = NULL, .val = 's' },
    { .name = "global", .has_arg = 0, .flag = NULL, .val = 1 },
    { .name = "io", .has_arg = 0, .flag = NULL, .val = 10 },
    { .name = "gpu", .has_arg = 1, .flag = NULL, .val = 'g' },
    { .name = "cuda-device", .has_arg = 1, .flag = NULL, .val = 'g' },
    { .name = "cuda-dev", .has_arg = 1, .flag = NULL, .val = 'g' },
    { .name = "dev", .has_arg = 1, .flag = NULL, .val = 'g' },
    { .name = "type", .has_arg = 1, .flag = NULL, .val = 't' },
    { .name = "mode", .has_arg = 1, .flag = NULL, .val = 't' },
    { .name = "bench", .has_arg = 1, .flag = NULL, .val = 't' },
    { .name = "benchmark", .has_arg = 1, .flag = NULL, .val = 't' },
    { .name = "test", .has_arg = 1, .flag = NULL, .val = 't' },
    { .name = "count", .has_arg = 1, .flag = NULL, .val = 'c' },
    { .name = "verbose", .has_arg = 0, .flag = NULL, .val = 'v' },
    { .name = "si", .has_arg = 0, .flag = NULL, .val = 2 },
    { .name = "vec", .has_arg = 1, .flag = NULL, .val = 'V'},
    { .name = "len", .has_arg = 1, .flag = NULL, .val = 'L' },
    { .name = "help", .has_arg = 0, .flag = NULL, .val = 'h' },
    { .name = NULL, .has_arg = 0, .flag = NULL, .val = 0 }
};


/* List supported benchmark types */
static void list_bench_modes()
{
    fprintf(stderr, "Benchmark types\n");
    fprintf(stderr, "  %-18s  %-56s\n", "name", "explanation");

    const bench_mode_t* mode = all_benchmarking_modes;

    while (*mode != BENCH_DO_NOTHING)
    {
        fprintf(stderr, "  %-18s  %-56s\n", bench_mode_name(*mode), bench_mode_desc(*mode));
        ++mode;
    }
    fprintf(stderr, "\n");
}


/* List local GPUs */
static void list_gpu_devices(size_t size_factor)
{
    // TODO rewrite this to use gpu_info instead and extend gpu_info_t
    hipError_t err;

    int gpu_device_count = 0;
    err = hipGetDeviceCount(&gpu_device_count);
    if (err != hipSuccess)
    {
        log_error("Unexpected error: %s", hipGetErrorString(err));
        gpu_device_count = 0;
    }

    fprintf(stderr, "Devices\n");
    fprintf(stderr, "  %2s %-20s %-9s %-8s %3s %7s %3s %4s   %-13s\n",
            "id", "name", "io addr", "map hmem", "tcc", "unified", "L1", "rdma", "glob mem size");

    for (int i = 0; i < gpu_device_count; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            log_error("Unexpected error: %s", hipGetErrorString(err));
            exit('h');
        }

        int rdma = !strncmp("Tesla", prop.name, 5) || !strncmp("Quadro", prop.name, 6);

        fprintf(stderr, "  %2d %-20s %02x:%02x.%-3x",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, 0);
        
        fprintf(stderr, " %8s", prop.canMapHostMemory ? "yes" : "no");
        fprintf(stderr, " %3s", prop.tccDriver ? "yes" : "no");
        fprintf(stderr, " %7s", prop.unifiedAddressing ? "yes" : "no");
        fprintf(stderr, " %3s", prop.globalL1CacheSupported ? "yes" : "no");
        fprintf(stderr, " %4s", rdma ? "yes" : "no");

        fprintf(stderr, "   %9.02f %-3s", prop.totalGlobalMem / (double) size_factor, 
                size_factor == 1e3 ? "kB" : "KiB");

        fprintf(stderr, "\n");
    }
    fprintf(stderr, "\n");
}


/* Retrieve local cluster node ID */
static unsigned get_local_node_id(unsigned adapter_no)
{
    sci_error_t err = SCI_ERR_OK;
    unsigned node_id = NO_NODE;

    sci_desc_t desc;
    SCIOpen(&desc, 0, &err);
    if (err != SCI_ERR_OK)
    {
        log_error("%s", SCIGetErrorString(err));
        exit(1);
    }

    SCIGetLocalNodeId(adapter_no, &node_id, 0, &err);
    if (err != SCI_ERR_OK)
    {
        log_error("%s", SCIGetErrorString(err));
        exit(1);
    }

    SCIClose(desc, 0, &err);

    return node_id;
}


/* Give program usage */
static void give_usage(const char* progname)
{
    fprintf(stderr,
            "Usage: %s [--size=<size>]\n"
            "   or: %s --rn=<remote node id> [--bench=<benchmark type>] [--size=<size>]\n"
            "\nDescription\n"
            "    Benchmark how long it takes to transfer memory between a local and a\n"
            "    remote segment across an NTB link.\n"
            "\nClient mode arguments\n"
            "  --bench=<bencmark type>  specify benchmark type, default is dma-push\n"
            "  --rn=<node id>           remote node identifier\n"
            "  --rseg=<segment id>      number identifying the remote segment\n"
            "  --count=<number>         number of times to repeat test (defaults to 1)\n"
            "\nDMA vector options (client mode)\n"
            "  --vec=<number>           divide segment into a number of DMA vector elements (defaults to 1)\n"
            "  --len=<number>           repeat the entire vector a number of times (defaults to 1)\n"
            "\nOptional arguments (both client and server mode)\n"
            "  --size=<size>            memory size in KiB (or kB if --si is set), default is %u KiB\n"
            "  --global                 create local segment with SCI_FLAG_DMA_GLOBAL\n"
            "  --io                     map remote segment with SCI_FLAG_IO_MAP_IOSPACE\n"
            "  --adapter=<adapter no>   local host adapter card number (defaults to 0)\n"
            "  --lseg=<segment id>      number identifying the local segment\n"
            "  --gpu=<gpu id>           specify a local GPU (if not given, buffer is allocated in RAM)\n"
            "  --verbose                increase verbosity level\n"
            "  --si                     use SI units (1000s) instead of IEC units (1024s)\n"
            "  --help                   show list of local GPUs and benchmark types\n"
            "\n"
            , progname, progname, DEFAULT_LOCAL_SEGMENT_COUNT);
}


int main(int argc, char** argv)
{
    /* Program parameters */
    unsigned local_adapter = 0;
    unsigned remote_node_id = NO_NODE;
    unsigned local_node_id = NO_NODE;
    unsigned local_segment_id = NO_ID;
    unsigned remote_segment_id = NO_ID;
    int local_gpu_id = NO_GPU;

    size_t local_segment_count = DEFAULT_LOCAL_SEGMENT_COUNT;
    size_t local_segment_factor = 1 << 10;

    size_t vec_div = 1;
    size_t vec_len = 1;

    size_t repeat_count = 1;
    bench_mode_t mode = BENCH_DMA_PUSH_TO_REMOTE;

    int global = 0;
    int io = 0;
    

    /* Parse program arguments */
    int opt, idx;
    char* str;

    while ((opt = getopt_long(argc, argv, "-:a:n:l:r:s:g:m:c:vV:L:h", options, &idx)) != -1)
    {
        switch (opt)
        {
            case ':': // missing value
                log_error("Option %s requires a value", argv[optind-1]);
                give_usage(argv[0]);
                exit(':');

            case '?': // unknown option
                log_error("Unknown option: %s", argv[optind-1]);
                give_usage(argv[0]);
                exit('?');

            case 'h': // show help
                list_gpu_devices(local_segment_factor);
                list_bench_modes();
                fprintf(stderr, "\n");
                give_usage(argv[0]);
                exit('h');

            case 'a': // set local adapter number
                str = NULL;
                local_adapter = strtoul(optarg, &str, 10);
                if (str == NULL || *str != '\0')
                {
                    log_error("Argument --adapter must be a valid adapter number");
                    exit('a');
                }
                break;

            case 'n': // set remote node
                str = NULL;
                remote_node_id = strtoul(optarg, &str, 10);
                if (str == NULL || *str != '\0' || remote_node_id == NO_NODE)
                {
                    log_error("Argument --remote-node must be a valid node identifier");
                    exit('n');
                }
                break;

            case 'l': // set local segment ID
                str = NULL;
                local_segment_id = strtoul(optarg, &str, 16);
                if (str == NULL || *str != '\0')
                {
                    log_error("Argument --local-id must be a valid segment identifier");
                    exit('l');
                }
                break;

            case 'r': // set remote segment ID
                str = NULL;
                remote_segment_id = strtoul(optarg, &str, 16);
                if (str == NULL || *str != '\0')
                {
                    log_error("Argument --remote-id must be a valid segment identifier");
                    exit('r');
                }
                break;

            case 's': // set segment size
                str = NULL;
                local_segment_count = strtoul(optarg, &str, 0);
                if (str == NULL || *str != '\0')
                {
                    log_error("Argument --size must be a valid segment size in %s", local_segment_factor == 1e3 ? "kB" : "KiB");
                    exit('s');
                }
                else if (local_segment_count == 0)
                {
                    local_segment_count = DEFAULT_LOCAL_SEGMENT_COUNT;
                    log_debug("Setting size to %u %s", local_segment_count, local_segment_factor == 1e3 ? "kB" : "KiB");
                }
                break;

            case 'g': // set local GPU
                str = NULL;
                local_gpu_id = strtol(optarg, &str, 10);
                if (str == NULL || *str != '\0' || local_gpu_id < 0)
                {
                    log_error("Argument --gpu must be a valid GPU number");
                    exit('g');
                }
                break;

            case 't': // set benchmark type
                mode = bench_mode_from_name(optarg);
                if (mode == BENCH_DO_NOTHING)
                {
                    log_error("Argument --type must be a valid benchmark type, see --help for a list of valid types");
                    exit('m');
                }
                break;

            case 'c': // set repeat count
                str = NULL;
                repeat_count = strtoul(optarg, &str, 10);
                if (str == NULL || *str != '\0' || repeat_count == 0)
                {
                    log_error("Argument --count must be at least 1");
                    exit('c');
                }
                break;

            case 'v': // increase verbosity
                ++verbosity;
                break;

            case 2: // use SI units instead of IEC units
                log_debug("Using SI units");
                local_segment_factor = 1e3;
                break;

            case 'V': // set DMA vector element length
                str = NULL;
                vec_div = strtoul(optarg, &str, 0);
                if (str == NULL || *str != '\0' || vec_div == 0)
                {
                    log_error("Argument --vec must be at least 1");
                    exit('V');
                }
                break;

            case 'L': // set DMA vector length
                str = NULL;
                vec_len = strtoul(optarg, &str, 10);
                if (str == NULL || *str != '\0' || vec_len == 0)
                {
                    log_error("Argument --len must be at least 1");
                    exit('L');
                }
                break;

            case 1:
                global = 1;
                break;

            case 10:
                io = 1;
                break;
        }
    }

    /* Sanity checking */
    if (remote_node_id == NO_NODE && local_segment_count == 0)
    {
        log_error("Either segment size or remote node identifier must be specified");
        give_usage(argv[0]);
        exit(1);
    }
    if (remote_node_id != NO_NODE && mode == BENCH_DO_NOTHING)
    {
        log_error("No benchmark type is specified");
        give_usage(argv[0]);
        exit(1);
    }

    /* Get number of CUDA enabled GPUs */
    if (local_gpu_id != NO_GPU)
    {
        int gpu_count;
        if (hipGetDeviceCount(&gpu_count) != hipSuccess)
        {
            log_warn("Failed to initialize CUDA, setting GPU will not work");
            gpu_count = 0;
        }

        if (local_gpu_id >= gpu_count)
        {
            log_error("Invalid local GPU selected");
            exit(1);
        }
    }

    /* Initialize SISCI API */
    sci_error_t err = SCI_ERR_OK;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        log_error("%s", SCIGetErrorString(err));
        exit(1);
    }

    local_node_id = get_local_node_id(local_adapter);
    log_debug("Local node ID %u (adapter %u)", local_node_id, local_adapter);

    if (local_gpu_id != NO_GPU)
    {
        log_info("GPU segment is specified");
    }

    /* Run as client or server */
    if (remote_node_id == NO_NODE)
    {
        log_info("Segment size is set to %lu %s", local_segment_count, local_segment_factor == 1e3 ? "kB" : "KiB");

        if (mode != BENCH_DO_NOTHING && mode != BENCH_DMA_PUSH_TO_REMOTE)
        {
            log_warn("Setting benchmark type has no effect in server mode");
        }

        if (repeat_count != 1)
        {
            log_warn("Setting benchmark repeat count has no effect in server mode");
        }

        if (vec_len != 1 || vec_div != 1)
        {
            log_warn("DMA vector options have no effect in server mode");
        }

        if (local_segment_id == NO_ID)
        {
            log_info("No local segment ID specified, using local node ID...");
            local_segment_id = local_node_id;
        }
        log_info("Connect to node %u with segment ID %x", local_node_id, local_segment_id);

        server(local_adapter, local_gpu_id, local_segment_id, local_segment_count * local_segment_factor, global);
    }
    else
    {
        if (local_segment_id == NO_ID)
        {
            log_info("No local segment ID specified, using local node ID...");
            local_segment_id = local_node_id;
        }
        if (remote_segment_id == NO_ID)
        {
            log_info("No remote segment ID specified using remote node ID...");
            remote_segment_id = remote_node_id;
        }

        if (mode == BENCH_SCIMEMWRITE_TO_REMOTE && local_gpu_id != NO_GPU)
        {
            log_error("%s specified, but GPU buffer is selected", bench_mode_name(mode));
            SCITerminate();
            exit(1);
        }

        if ((vec_div != 1 || vec_len != 1) && !BENCH_IS_DMA(mode))
        {
            log_warn("DMA vector options have no effect when benchmark type is not DMA");
            vec_div = vec_len = 1;
        }

        log_info("Initializing transfer list...");

        translist_t ts;
        if (translist_create(&ts, local_adapter, local_segment_id, local_segment_count * local_segment_factor, remote_node_id, remote_segment_id, local_gpu_id, global) != 0)
        {
            log_error("Unexpected error when creating transfer list, aborting...");
            SCITerminate();
            exit(1);
        }

        translist_desc_t tsd = translist_desc(ts);
        size_t segment_size = tsd.segment_size;
        log_info("Transfer segment size %.2f %s", segment_size / (double) local_segment_factor, local_segment_factor == 1e3 ? "kB" : "KiB");

        if (vec_div >= segment_size || segment_size / vec_div == 0)
        {
            log_error("Number of DMA vector entries is larger than segment size");
            translist_delete(ts);
            SCITerminate();
            exit(1);
        }
    
        /* Fill transfer list */
        size_t entry_size = segment_size / vec_div;

        for (size_t k = 0; k < vec_len; ++k)
        {
            for (size_t v = 0; v < vec_div; ++v)
            {
                if (translist_insert(ts, v * entry_size, v * entry_size, entry_size) != 0)
                {
                    log_error("Failed to create transfer list");
                    break;
                }
            }
        }

        /* Create and run benchmark */
        bench_t bench_conf = {
            .benchmark_mode = mode,
            .num_runs = repeat_count,
            .transfer_list = ts
        };

        result_t* result = (result_t*) malloc(sizeof(result_t) + sizeof(uint64_t) * repeat_count);
        if (result == NULL)
        {
            log_error("Out of resources");
            translist_delete(ts);
            SCITerminate();
            exit(1);
        }

        if (client(local_adapter, &bench_conf, result, io ? SCI_FLAG_IO_MAP_IOSPACE : 0) == 0)
        {
            log_info("Total runtime is %.2f s", result->total_runtime / 1e6l);
            log_info("Avg bandwidth is %.2f %-5s", (double) (result->total_size * repeat_count) / (double) result->total_runtime, local_segment_factor == 1e3 ? "MB/s" : "MiB/s");
            report_summary(stdout, &bench_conf, result, local_segment_factor != 1e3);
            report_bandwidth(stdout, &bench_conf, result, local_segment_factor != 1e3);
        }
        else
        {
            log_warn("Benchmark failed, skipping results");
        }

        free(result);
        translist_delete(ts);
    }

    SCITerminate();
    exit(0);
}
