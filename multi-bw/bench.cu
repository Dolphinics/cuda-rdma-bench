#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include <cstdio>
#include "bench.h"
#include "devbuf.h"
#include "hostbuf.h"
#include "stream.h"
#include "event.h"

using std::vector;
using std::runtime_error;
using std::string;


static string bytesToUnit(size_t size)
{
    char buffer[1024];
    const char* units[] = { "B  ", "KiB", "MiB", "GiB", "TiB" };
    size_t i = 0, n = sizeof(units) / sizeof(units[0]);

    double csize = (double) size;

    while (i < (n - 1) && csize >= 1024.0)
    {
        csize /= 1024.0;
        ++i;
    }

    snprintf(buffer, sizeof(buffer), "%.2f %s", csize, units[i]);
    return string(buffer);
}


static string transferDirectionToString(hipMemcpyKind direction)
{
    if (direction == hipMemcpyHostToDevice)
    {
        return string("HtoD");
    }
    if (direction == hipMemcpyDeviceToHost)
    {
        return string("DtoH");
    }

    return string("unknown");
}


static void timeTransfers(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        hipStream_t stream = *spec.cudaStream;
        const DeviceBufferPtr& deviceBuffer = spec.deviceBuffer;
        const HostBufferPtr& hostBuffer = spec.hostBuffer;

        const void* src = spec.direction == hipMemcpyDeviceToHost ? deviceBuffer->buffer : hostBuffer->buffer;
        void* dst = spec.direction == hipMemcpyDeviceToHost ? hostBuffer->buffer : deviceBuffer->buffer;
        size_t size = min(deviceBuffer->length, hostBuffer->length); 

        err = hipEventRecord(spec.cudaEvents->started, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipMemcpyAsync(dst, src, size, spec.direction, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventRecord(spec.cudaEvents->stopped, stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


static void syncStreams(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    for (const TransferSpec& spec : transferSpecs)
    {
        err = hipStreamSynchronize(*spec.cudaStream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }
}


void runBandwidthTest(const vector<TransferSpec>& transferSpecs)
{
    hipError_t err;

    // Create timing events on the null stream
    TimingDataPtr nullStreamTiming = createTimingData();
    err = hipEventRecord(nullStreamTiming->started);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    // Execute transfers
    try
    {
        fprintf(stdout, "Executing transfers.....");
        fflush(stdout);
        timeTransfers(transferSpecs);
        fprintf(stdout, "DONE\n");
        fflush(stdout);
    }
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }

    // Synchronize all streams
    try
    {
        fprintf(stdout, "Synchronizing streams...");
        fflush(stdout);

        syncStreams(transferSpecs);

        err = hipEventRecord(nullStreamTiming->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventSynchronize(nullStreamTiming->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        fprintf(stdout, "DONE\n");
        fflush(stdout);
    } 
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }


    // Print results
    fprintf(stdout, "\n");
    fprintf(stdout, "=====================================================================================\n");
    fprintf(stdout, " %2s   %-15s   %13s   %-8s   %-12s   %-10s\n",
            "ID", "Device name", "Transfer size", "Direction", "Time elapsed", "Bandwidth");
    fprintf(stdout, "-------------------------------------------------------------------------------------\n");
    fflush(stdout);

    size_t totalSize = 0;
    double aggrElapsed = .0;
    double timedElapsed = nullStreamTiming->usecs();

    for (const TransferSpec& res : transferSpecs)
    {
        size_t size = min(res.deviceBuffer->length, res.hostBuffer->length);
        double elapsed = res.cudaEvents->usecs();
        double bandwidth = (double) size / elapsed;

        totalSize += size;
        aggrElapsed += elapsed;

        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, res.deviceBuffer->device);
        if (err != hipSuccess)
        {
            prop.name[0] = 'E';
            prop.name[1] = 'R';
            prop.name[2] = 'R';
            prop.name[3] = '!';
            prop.name[4] = '\0';
        }

        fprintf(stdout, " %2d   %-15s   %13s    %8s   %9.0f µs    %10.2f MiB/s \n",
                res.deviceBuffer->device, 
                prop.name, 
                bytesToUnit(size).c_str(), 
                transferDirectionToString(res.direction).c_str(),
                elapsed,
                bandwidth
               );
        fflush(stdout);
    }
    fprintf(stdout, "=====================================================================================\n");

    fprintf(stdout, "\n");
    fprintf(stdout, "Aggregated total time      : %12.0f µs\n", aggrElapsed);
    fprintf(stdout, "Aggregated total bandwidth : %12.2f MiB/s\n", (double) totalSize / aggrElapsed);
    fprintf(stdout, "Estimated elapsed time     : %12.0f µs\n", timedElapsed);
    fprintf(stdout, "Timed total bandwidth      : %12.2f MiB/s\n", (double) totalSize / timedElapsed);
    fprintf(stdout, "\n");
    fflush(stdout);
}
