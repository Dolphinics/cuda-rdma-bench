#include <hip/hip_runtime.h>
#include <vector>
#include <cstdio>
#include <exception>
#include <stdexcept>
#include <string>
#include <cstring>
#include "bench.h"
#include "devbuf.h"
#include "hostbuf.h"

using std::vector;
using std::runtime_error;
using std::string;


struct StreamData
{
    int          device;
    void*        buffer;
    size_t       length;
    hipStream_t stream;
    hipEvent_t  started;
    hipEvent_t  stopped;
};


static string bytesToUnit(size_t size)
{
    char buffer[1024];
    const char* units[] = { "B  ", "KiB", "MiB", "GiB", "TiB" };
    size_t i = 0, n = sizeof(units) / sizeof(units[0]);

    double csize = (double) size;

    while (i < (n - 1) && csize >= 1024.0)
    {
        csize /= 1024.0;
        ++i;
    }

    snprintf(buffer, sizeof(buffer), "%.2f %s", csize, units[i]);
    return string(buffer);
}


static double usecsElapsed(const StreamData& data)
{
    float milliseconds = .0f;

    hipError_t err = hipEventElapsedTime(&milliseconds, data.started, data.stopped);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    return (double) milliseconds * 1000;
}


static void measureMemcpyBandwidth(void* hostBuffer, vector<StreamData>& streamData, hipMemcpyKind kind)
{
    hipError_t err;

    // Start transfers
    for (vector<StreamData>::iterator it = streamData.begin(); it != streamData.end(); ++it)
    {
        const void* src = kind == hipMemcpyDeviceToHost ? it->buffer : hostBuffer;
        void* dst = kind == hipMemcpyDeviceToHost ? hostBuffer : it->buffer;

        err = hipEventRecord(it->started, it->stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipMemcpyAsync(dst, src, it->length, kind, it->stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventRecord(it->stopped, it->stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }
    }

    // Print results
    size_t totalSize = 0;
    double totalTime = 0;
    for (vector<StreamData>::iterator it = streamData.begin(); it != streamData.end(); ++it)
    {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, it->device);
        if (err != hipSuccess)
        {
            fprintf(stderr, "WARNING: %s\n", hipGetErrorString(err));
            prop.name[0] = '\0';
        }

        // make sure stream is done
        err = hipEventSynchronize(it->stopped);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        double usecs = usecsElapsed(*it);
        double bandwidth = (double) it->length / usecs;

        totalSize += it->length;
        totalTime += usecs;

        fprintf(stdout, "%4d %-25s %10s %8.0f µs %10.2f MiB/s\n",
                it->device, 
                prop.name, 
                bytesToUnit(it->length).c_str(), 
                usecs,
                bandwidth
               );
    }
}


static void runBandwidthTest(const HostBuffer& hostBuffer, const vector<DeviceBuffer>& deviceBuffers, hipMemcpyKind kind)
{
    hipError_t err;

    // Create streams and events
    vector<StreamData> streamData;
    for (vector<DeviceBuffer>::const_iterator it = deviceBuffers.begin(); it != deviceBuffers.end(); ++it)
    {
        StreamData data;
        data.device = it->device;
        data.buffer = it->buffer;
        data.length = it->length;
        
        err = hipStreamCreate(&data.stream);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventCreate(&data.started);
        if (err != hipSuccess)
        {
            hipStreamDestroy(data.stream);
            throw runtime_error(hipGetErrorString(err));
        }

        err = hipEventCreate(&data.stopped);
        if (err != hipSuccess)
        {
            hipStreamDestroy(data.stream);
            hipEventDestroy(data.started);
            throw runtime_error(hipGetErrorString(err));
        }

        streamData.push_back(data);
    }

    // Run measurements
    try
    {
        measureMemcpyBandwidth(hostBuffer.buffer, streamData, kind);
    }
    catch (const runtime_error& e)
    {
        fprintf(stderr, "Unexpected error, aborting...\n");
    }

    // Clean up
    for (vector<StreamData>::iterator it = streamData.begin(); it != streamData.end(); ++it)
    {
        hipEventDestroy(it->started);
        hipEventDestroy(it->stopped);
        hipStreamDestroy(it->stream);
    }
}


void benchmark(const vector<HostBuffer>& buffers, const vector<int>& devices, const vector<hipMemcpyKind>& modes)
{
    // TODO: Print out some info
    // #stream: x

    for (vector<hipMemcpyKind>::const_iterator kindIt = modes.begin(); kindIt != modes.end(); ++kindIt)
    {
        hipMemcpyKind kind = *kindIt;

        for (vector<HostBuffer>::const_iterator bufIt = buffers.begin(); bufIt != buffers.end(); ++bufIt)
        {
            // Get host buffer
            const HostBuffer& buffer = *bufIt;

            // Create device buffers
            vector<DeviceBuffer> deviceBuffers;
            for (vector<int>::const_iterator devIt = devices.begin(); devIt != devices.end(); ++devIt)
            {
                deviceBuffers.push_back(DeviceBuffer(*devIt, buffer.length));
            }

            // Run bandwidth test
            fprintf(stdout, "\n====================    %-14s   (%11s)    ====================\n",
                    kind == hipMemcpyDeviceToHost ? "DEVICE TO HOST" : "HOST TO DEVICE",
                    bytesToUnit(buffer.length).c_str()
                    );
            runBandwidthTest(buffer, deviceBuffers, kind);
        }
    }
}
