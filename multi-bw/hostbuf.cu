#include <hip/hip_runtime.h>
#include <memory>
#include <exception>
#include <stdexcept>
#include "hostbuf.h"


static void* createHostBuffer(size_t length, unsigned int flags)
{
    void* buffer;

    hipError_t err = hipHostAlloc(&buffer, length, flags);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return buffer;
}


HostBuffer::HostBuffer(size_t length, unsigned int flags)
    : length(length)
    , buffer(createHostBuffer(length, flags))
{
}


HostBuffer::~HostBuffer()
{
    hipHostFree(buffer);
}

