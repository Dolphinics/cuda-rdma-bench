#include <hip/hip_runtime.h>
#include <map>
#include <memory>
#include <exception>
#include <stdexcept>
#include "stream.h"

using std::runtime_error;
typedef std::map<int, StreamPtr> StreamMap;

static StreamMap streams;


static void deleteStream(hipStream_t* stream)
{
    hipStreamSynchronize(*stream);
    hipStreamDestroy(*stream);
    delete stream;
}


static StreamPtr createStream()
{
    hipStream_t* stream = new hipStream_t;
    hipError_t err = hipStreamCreateWithFlags(stream, hipStreamNonBlocking);
    //hipError_t err = hipStreamCreateWithFlags(stream, hipStreamDefault);
    if (err != hipSuccess)
    {
        delete stream;
        throw runtime_error(hipGetErrorString(err));
    }

    return StreamPtr(stream, &deleteStream);
}


StreamPtr retrieveStream(int device, StreamSharingMode sharing)
{
    if (sharing != perTransfer)
    {
        if (sharing == singleStream)
        {
            device = -1;
        }

        // Try to find stream in map
        StreamMap::iterator lowerBound = streams.lower_bound(device);
        if (lowerBound != streams.end() && !(streams.key_comp()(device, lowerBound->first)))
        {
            return lowerBound->second;
        }

        // Stream was not found in map, create it and return it
        StreamPtr stream = createStream();
        streams.insert(lowerBound, StreamMap::value_type(device, stream));
        return stream;
    }

    // Create a new stream every time
    return createStream();
}

