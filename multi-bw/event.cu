#include <hip/hip_runtime.h>
#include <memory>
#include <exception>
#include <stdexcept>
#include "event.h"


static void deleteTimingData(TimingData* data)
{
    hipEventDestroy(data->started);
    hipEventDestroy(data->stopped);
    delete data;
}


TimingDataPtr createTimingData()
{
    hipError_t err;

    TimingData* data = new TimingData;

    err = hipEventCreate(&data->started);
    if (err != hipSuccess)
    {
        delete data;
        throw std::runtime_error(hipGetErrorString(err));
    }

    err = hipEventCreate(&data->stopped);
    if (err != hipSuccess)
    {
        hipEventDestroy(data->started);
        delete data;
        throw std::runtime_error(hipGetErrorString(err));
    }

    return TimingDataPtr(data, &deleteTimingData);
}



double TimingData::usecs() const
{
    float milliseconds = .0f;

    hipError_t err = hipEventElapsedTime(&milliseconds, started, stopped);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return (double) milliseconds * 1000;
}
