#include <hip/hip_runtime.h>
#include <getopt.h>
#include <string>
#include <cstdlib>
#include <cstring>
#include <strings.h>
#include <cstdio>
#include <vector>
#include <exception>
#include <stdexcept>
#include "devbuf.h"
#include "hostbuf.h"
#include "bench.h"
#include "event.h"

using namespace std;


static void showUsage(const char* fname)
{
    fprintf(stderr, "Usage: %s --transfer=<transfer specs>... [--streams=<mode>] [--list] [--help]\n" 
            "\nDescription\n"
            "    This program uses multiple CUDA streams in an attempt at optimizing data\n"
            "    transfers between host and multiple CUDA devices using hipMemcpyAsync().\n"
            "\nProgram options\n"
            "  --streams=<mode>      stream modes for transfers\n"
            "  --list               list available CUDA devices and quit\n"
            "  --help               show this help text and quit\n"
            "\nStream modes\n"
            "  per-transfer         one stream per transfer [default]\n"
            "  per-device           transfers to the same device share streams\n"
            "  only-one             all transfers share the same single stream\n"
            "\nTransfer specification format\n"
            "    <device>[:<direction>][:<size>][:<memory options>...]\n"
            "\nTransfer specification arguments\n"
            "  <device>             CUDA device to use for transfer\n"
            "  <direction>          transfer directions\n"
            "  <size>               transfer size in bytes [default is 32 MiB]\n"
            "  <memory options>     memory allocation options\n"
            "\nTransfer directions\n"
            "  HtoD                 host to device transfer (RAM to GPU)\n"
            "  DtoH                 device to host transfer (GPU to RAM)\n"
            "  both                 first HtoD then DtoH [default]\n"
            "  reverse              first DtoH then HtoD\n"
            "\nMemory options format\n"
            "   option1,option2,option3,...\n"
            "\nMemory options\n"
            "  mapped               map host memory into CUDA address space\n"
            "  managed              allocate managed memory on the device\n"
            "  wc                   allocate write-combined memory on the host\n"
            "\n"
            ,
            fname
           );
}


static void listDevices()
{
    hipError_t err;

    int deviceCount = 0;
    err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        throw runtime_error(hipGetErrorString(err));
    }

    fprintf(stderr, "\n %2s   %-20s   %-9s   %8s   %8s   %8s   %2s\n",
            "ID", "Device name", "IO addr", "Managed", "Unified", "Mappable", "#");
    fprintf(stderr, "-----------------------------------------------------------------------------\n");
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        if (prop.computeMode == hipComputeModeProhibited)
        {
            continue;
        }

        fprintf(stderr, " %2d   %-20s   %02x:%02x.%-3x   %8s   %8s   %8s   %2d\n",
                i, prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID,
                prop.managedMemory ? "yes" : "no", 
                prop.unifiedAddressing ? "yes" : "no",
                prop.canMapHostMemory ? "yes" : "no",
                prop.asyncEngineCount);
    }
    fprintf(stderr, "\n");
}


static bool isValidDevice(int device)
{
    hipDeviceProp_t prop;

    hipError_t err = hipGetDeviceProperties(&prop, device);
    if (err != hipSuccess)
    {
        return false;
    }

    if (prop.computeMode == hipComputeModeProhibited)
    {
        return false;
    }

    return true;
}


static void parseDevice(vector<int>& devices, const char* token)
{
    if (strcasecmp("all", token) != 0)
    {
        char* strptr = NULL;
        int device = strtol(token, &strptr, 10);
        if (strptr == NULL || *strptr != '\0' || !isValidDevice(device))
        {
            fprintf(stderr, "Invalid transfer specification: '%s' is not a valid device\n", token);
            throw 3;
        }
        devices.push_back(device);
    }
    else
    {
        int deviceCount = 0;
        hipError_t err = hipGetDeviceCount(&deviceCount);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        for (int device = 0; device < deviceCount; ++device)
        {
            if (isValidDevice(device))
            {
                devices.push_back(device);
            }
        }
    }
}


static void parseDirection(vector<hipMemcpyKind>& directions, const char* token)
{
    if (strcasecmp("dtoh", token) == 0)
    {
        directions.push_back(hipMemcpyDeviceToHost);
    }
    else if (strcasecmp("htod", token) == 0)
    {
        directions.push_back(hipMemcpyHostToDevice);
    }
    else if (strcasecmp("both", token) == 0)
    {
        directions.push_back(hipMemcpyHostToDevice);
        directions.push_back(hipMemcpyDeviceToHost);
    }
    else if (strcasecmp("reverse", token) == 0)
    {
        directions.push_back(hipMemcpyDeviceToHost);
        directions.push_back(hipMemcpyHostToDevice);
    }
}


static void parseSize(size_t& size, const char* token)
{
    char* strptr = NULL;
    size = strtoul(token, &strptr, 0);
    if (strptr == NULL || *strptr != '\0')
    {
        size = 0;
    }
}


static void parseTransferSpecification(vector<TransferSpec>& transferSpecs, char* specStr)
{
    vector<int> devices;
    vector<hipMemcpyKind> directions;
    size_t size = 0;

    unsigned int hostAllocFlags = hipHostMallocDefault;
    unsigned int deviceAllocFlags = 0;
    bool useManagedDeviceMem = false;

    // First token must be device
    const char* delim = ":,";
    char* token = strtok(specStr, delim);
    parseDevice(devices, token);

    // The remaining of the transfer specification may be in arbitrary order
    // because we want to be nice
    while ((token = strtok(NULL, delim)) != NULL)
    {
        if (directions.empty())
        {
            parseDirection(directions, token);
        }

        if (strcasecmp("mapped", token) == 0)
        {
            hostAllocFlags |= hipHostMallocMapped;
        }
        else if (strcasecmp("write-combined", token) == 0 || strcasecmp("wc", token) == 0)
        {
            hostAllocFlags |= hipHostMallocWriteCombined;
        }
        else if (strcasecmp("managed", token) == 0)
        {
            useManagedDeviceMem = true;
        }

        if (size == 0)
        {
            parseSize(size, token);
        }
    }

    // Insert default values if necessary
    if (directions.empty())
    {
        directions.push_back(hipMemcpyHostToDevice);
        directions.push_back(hipMemcpyDeviceToHost);
    }
    if (size == 0)
    {
        size = 32 << 20;
    }

    // Try to allocate buffers and create transfer specification
    try
    {
        fprintf(stdout, "Allocating buffers......");
        fflush(stdout);

        for (hipMemcpyKind transferMode : directions)
        {
            for (int device : devices)
            {
                TransferSpec spec;
                spec.deviceBuffer = DeviceBufferPtr(new DeviceBuffer(device, size)); // FIXME: Managed memory
                spec.hostBuffer = HostBufferPtr(new HostBuffer(size, hostAllocFlags));
                spec.direction = transferMode;

                transferSpecs.push_back(spec);
            }
        }

        fprintf(stdout, "DONE\n");
        fflush(stdout);
    }
    catch (const runtime_error& e)
    {
        fprintf(stdout, "FAIL\n");
        fflush(stdout);
        throw e;
    }
}


static void parseArguments(int argc, char** argv, StreamSharingMode& streamMode, vector<TransferSpec>& transferSpecs)
{
    // Define program arguments
    const option opts[] = {
        { .name = "transfer", .has_arg = 1, .flag = NULL, .val = 't' },
        { .name = "streams", .has_arg = 1, .flag = NULL, .val = 's' },
        { .name = "list", .has_arg = 0, .flag = NULL, .val = 'l' },
        { .name = "help", .has_arg = 0, .flag = NULL, .val = 'h' },
        { .name = NULL, .has_arg = 0, .flag = NULL, .val = 0 }
    };

    // Parse arguments
    int opt, idx;
    while ((opt = getopt_long(argc, argv, "-:t:s:lh", opts, &idx)) != -1)
    {
        switch (opt)
        {
            case ':': // missing value
                fprintf(stderr, "Option %s requires a value\n", argv[optind-1]);
                throw 1;

            case '?': // unknown option
                fprintf(stderr, "Unknown option: %s\n", argv[optind-1]);
                throw 1;

            case 't': // transfer specification
                parseTransferSpecification(transferSpecs, optarg);
                break;

            case 's': // stream sharing mode
                if (strcasecmp("per-transfer", optarg) == 0)
                {
                    streamMode = perTransfer;
                }
                else if (strcasecmp("per-device", optarg) == 0 || strcasecmp("per-gpu", optarg) == 0)
                {
                    streamMode = perDevice;
                }
                else if (strcasecmp("only-one", optarg) == 0 || strcasecmp("single", optarg) == 0)
                {
                    streamMode = singleStream;
                }
                else
                {
                    fprintf(stderr, "Unknown stream mode: %s\n", optarg);
                    throw 2;
                }
                break;

            case 'l': // list devices
                listDevices();
                throw 0;

            case 'h': // show help
                showUsage(argv[0]);
                throw 0;
        }
    }
}


int main(int argc, char** argv)
{
    StreamSharingMode streamMode = perTransfer;
    vector<TransferSpec> transferSpecs;

    // Parse program arguments
    try 
    {
        parseArguments(argc, argv, streamMode, transferSpecs);
    }
    catch (const runtime_error& e)
    {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }
    catch (const int e) 
    {
        return e;
    }

    try
    {
        // No transfer specifications?
        if (transferSpecs.empty())
        {
            char buffer[64];
            snprintf(buffer, sizeof(buffer), "all");
            parseTransferSpecification(transferSpecs, buffer);
        }

        // Create streams and timing events
        for (TransferSpec& spec : transferSpecs)
        {
            spec.cudaStream = retrieveStream(spec.deviceBuffer->device, streamMode);
            spec.cudaEvents = createTimingData();
        }

        // Run bandwidth test
        runBandwidthTest(transferSpecs);
    }
    catch (const runtime_error& e)
    {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

    return 0;
}
