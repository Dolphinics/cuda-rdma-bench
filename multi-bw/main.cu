#include <hip/hip_runtime.h>
#include <getopt.h>
#include <cstdlib>
#include <cstring>
#include <cstdio>
#include <vector>
#include <exception>
#include <stdexcept>
#include "devbuf.h"
#include "hostbuf.h"
#include "bench.h"

using namespace std;


// Number of available CUDA devices
static int deviceCount = 0;

// What devices to use for the bandwidth test
static vector<int> devices;

// Different host buffers to use for the bandwidth test
static vector<HostBuffer> buffers;

// Different copy modes to use for the bandwidth test
static vector<hipMemcpyKind> modes;

// Specifies that a single CUDA stream should be used
static int singleStream = 0;

// Specifies that devices that are specified multiple times should share
// the same stream
static int shareStream = 0;


static void showUsage(const char* fname)
{
    fprintf(stderr, "Usage: %s --device=<id>... --size=<size>... [options]\n" 
            "\nDescription\n"
            "    As the CUDA samples bandwidthTest might not be able to fully utilize the bus,\n"
            "    this programs starts multiple hipMemcpyAsync transfers using multiple streams\n"
            "    in order to measure the maximum bandwidth.\n"
            "\nArguments\n"
            "  --size=<size>        transfer size in bytes\n"
            "  --device=<id | all>  specify CUDA device to use for transfer\n"
            "\nOptional arguments\n"
            "  --dtoh               specify device to host transfer (GPU to RAM)\n"
            "  --htod               specify host to device transfer (RAM to GPU)\n" 
            "  --mapped             map host memory into CUDA address space\n"
            "  --wc                 allocate write-combined host memory\n" 
            "  --single             use a single CUDA stream for all transfers\n"
            "  --share              devices specified multiple times share stream\n"
            "  --list               list available CUDA devices\n"
            "  --help               show this help\n"
            "\nNOTE: The arguments --size and --device can be can be specified multiple times\n"
            "        in order to test transferring different sizes and devices.\n",
            fname);
}


static void listDevices()
{
    hipError_t err;

    fprintf(stderr, "Available devices\n");
    for (int i = 0; i < deviceCount; ++i)
    {
        hipDeviceProp_t prop;

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            throw runtime_error(hipGetErrorString(err));
        }

        if (prop.computeMode == hipComputeModeProhibited)
        {
            continue;
        }

        fprintf(stderr, "  %2d %-25s %02x:%02x.%-3x\n",
                i, prop.name, prop.pciBusID, prop.pciDomainID, prop.pciDeviceID);
    }
    fprintf(stderr, "\n");
}


static void parseArguments(int argc, char** argv)
{
    vector<size_t> sizes;
    unsigned int flags = hipHostMallocDefault;

    // Define program arguments
    option opts[] = {
        { .name = "device", .has_arg = 1, .flag = NULL, .val = 'd' },
        { .name = "dev", .has_arg = 1, .flag = NULL, .val = 'd' },
        { .name = "size", .has_arg = 1, .flag = NULL, .val = 's' },
        { .name = "length", .has_arg = 1, .flag = NULL, .val = 's' },
        { .name = "len", .has_arg = 1, .flag = NULL, .val = 's' },
        { .name = "dtoh", .has_arg = 0, .flag = NULL, .val = hipMemcpyDeviceToHost },
        { .name = "htod", .has_arg = 0, .flag = NULL, .val = hipMemcpyHostToDevice },
        { .name = "mapped", .has_arg = 0, .flag = NULL, .val = 'm' },
        { .name = "write-combined", .has_arg = 0, .flag = NULL, .val = 'c' },
        { .name = "wc", .has_arg = 0, .flag = NULL, .val = 'c' },
        { .name = "single", .has_arg = 0, .flag = &singleStream, .val = 1 },
        { .name = "shared", .has_arg = 0, .flag = &shareStream, .val = 1 },
        { .name = "list", .has_arg = 0, .flag = NULL, .val = 'l' },
        { .name = "help", .has_arg = 0, .flag = NULL, .val = 'h' },
        { .name = NULL, .has_arg = 0, .flag = NULL, .val = 0 }
    };

    // Parse arguments
    int opt, idx;
    while ((opt = getopt_long(argc, argv, "-:d:s:mwcslh", opts, &idx)) != -1)
    {
        switch (opt)
        {
            case ':': // missing value
                fprintf(stderr, "Option %s requires a value\n", argv[optind-1]);
                throw 1;

            case '?': // unknown option
                fprintf(stderr, "Unknown option: %s\n", argv[optind-1]);
                throw 1;
    
            case 'd': // append device to device list
                {
                    if (strcmp(optarg, "all") == 0)
                    {
                        for (int i = 0; i < deviceCount; ++i)
                        {
                            devices.push_back(i);
                        }
                        break;
                    }

                    char* str = NULL;
                    int device = strtol(optarg, &str, 10);
                    if (str == NULL || *str != '\0' || device < 0 || device >= deviceCount)
                    {
                        throw "Argument --device must be a valid CUDA device";
                    }
                    devices.push_back(device);
                }
                break;

            case 's': // append transfer size to size list
                {
                    char* str = NULL;
                    size_t size = strtoull(optarg, &str, 0);
                    if (str == NULL || *str != '\0' || size == 0)
                    {
                        throw "Argument --size must be a valid byte count";
                    }
                    sizes.push_back(size);
                }
                break;

            case hipMemcpyDeviceToHost: // device to host
            case hipMemcpyHostToDevice: // host to device
                modes.push_back((hipMemcpyKind) opt);
                break;

            case 'm': // mapped memory
                flags |= hipHostMallocMapped;
                break;

            case 'c': // write combined memory
            case 'w':
                flags |= hipHostMallocWriteCombined;
                break;

            case 'l': // list devices
                listDevices();
                throw 0;

            case 'h': // show help
                showUsage(argv[0]);
                throw 0;
        }
    }

    if (modes.empty())
    {
        modes.push_back(hipMemcpyHostToDevice);
        modes.push_back(hipMemcpyDeviceToHost);
    }

    if (sizes.empty())
    {
        fprintf(stderr, "NOTE: No size argument given, using default size 32 MiB\n");
        sizes.push_back(32 << 20);
    }

    if (devices.empty())
    {
        fprintf(stderr, "NOTE: No devices specified, using all devices\n");
        for (int i = 0; i < deviceCount; ++i)
        {
            devices.push_back(i);
        }
    }

    // FIXME: Check if specified devices are allowed to use

    // Create host buffers
    for (vector<size_t>::const_iterator sizeIt = sizes.begin(); sizeIt != sizes.end(); ++sizeIt)
    {
        const size_t size = *sizeIt;
        buffers.push_back(HostBuffer(size, flags));
    }
}



int main(int argc, char** argv)
{
    // Find maximum GPU count
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Unexpected error: %s\n", hipGetErrorString(err));
        return 'd';
    }

    // Parse program arguments
    try 
    {
        parseArguments(argc, argv);
    }
    catch (const runtime_error& e)
    {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }
    catch (const int e) // FIXME: Hack
    {
        return e;
    }
    catch (const char* e) // FIXME: Hack
    {
        fprintf(stderr, "%s\n", e);
        return 1;
    }

    // Run bandwidth benchmark
    try
    {
        benchmark(buffers, devices, modes, shareStream, singleStream);
    }
    catch (const runtime_error& e)
    {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

    // Reset devices to supress warnings from cuda-memcheck
    for (vector<int>::const_iterator deviceIt = devices.begin(); deviceIt != devices.end(); ++deviceIt)
    {
        hipSetDevice(*deviceIt);
        hipDeviceReset();
    }

    return 0;
}
