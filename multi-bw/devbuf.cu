#include <hip/hip_runtime.h>
#include <memory>
#include <exception>
#include <stdexcept>
#include "devbuf.h"


static void* createDeviceBuffer(int device, size_t length)
{
    hipError_t err;

    err = hipSetDevice(device);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    void* buffer;
    err = hipMalloc(&buffer, length);
    if (err != hipSuccess)
    {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return buffer;
}


DeviceBuffer::DeviceBuffer(int device, size_t length)
    : device(device)
    , length(length)
    , buffer(createDeviceBuffer(device, length))
{ 
}


DeviceBuffer::~DeviceBuffer()
{
    hipFree(buffer);
}
