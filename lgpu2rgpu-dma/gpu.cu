#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sisci_api.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "gpu.h"
#include "reporting.h"


__global__ void gpu_memset_kernel(void* buf, size_t len, uint8_t val)
{
    const int num = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int pos = y * (gridDim.x * blockDim.x) + x;

    uint8_t* ptr = (uint8_t*) buf;

    for (size_t i = pos * (len / num), n = (pos + 1) * (len / num); i < n && i < len; ++i)
    {
        ptr[i] = val;
    }

    __syncthreads();
}


__global__ void gpu_memcmp_kernel(void* local, void* remote, size_t len, uint8_t* result)
{
    int num = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pos = y * (gridDim.x * blockDim.x) + x;

    uint8_t* l_ptr = (uint8_t*) local;
    uint8_t* r_ptr = (uint8_t*) remote;

    size_t i = pos * (len / num);
    size_t n = (pos + 1) * (len / num);

    for ( ; i < n && i < len && l_ptr[i] == r_ptr[i]; ++i);

    __syncthreads();

    result[pos] = i == n;
}


extern "C"
void gpu_memset(int gpu, void* ptr, size_t len, uint8_t val)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    dim3 grid;
    grid.x = 4;
    grid.y = 4;

    dim3 block;
    block.x = 4;
    block.y = 4;

    log_debug("Filling buffer with value %02x...", val);

    gpu_memset_kernel<<<grid, block>>>(ptr, len, val);

    hipDeviceSynchronize();
}


// Copy remote buffer to GPU buffer and do memcmp in parallel
int gpu_memcmp_gpu(int gpu, void* local, void* remote, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    dim3 grid;
    grid.x = 4;
    grid.y = 4;

    dim3 block;
    block.x = 4;
    block.y = 4;

    uint8_t* result = NULL;
    err = hipHostAlloc(&result, 32, hipHostMallocMapped);
    if (err != hipSuccess)
    {
        log_error("Out of resources: %s", hipGetErrorString(err));
        hipHostFree(result);
        return 0;
    }

    size_t i = 0, n = 256;
    for ( ; i < n; ++i)
    {
        result[i] = 0;
    }

    log_debug("Comparing local GPU memory %p to copied memory %p", local, remote);
    gpu_memcmp_kernel<<<grid, block>>>(local, remote, len, result);

    hipDeviceSynchronize();

    for (i = 0; i < n && result[i] != 0; ++i);

    hipHostFree(result);
    return i != n;
}


// Copy GPU-bound buffer to RAM and do regular memcmp
int gpu_memcmp_ram(int gpu, void* gpuptr, volatile void* ramptr, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    uint8_t* buf = NULL;
    err = hipHostAlloc(&buf, len, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate host memory: %s", hipGetErrorString(err));
        return 0;
    }

    err = hipMemcpy(buf, gpuptr, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    log_debug("Comparing local GPU memory %p to remote memory %p", gpuptr, ramptr);
    int equality = memcmp(buf, (void*) ramptr, len);

    hipHostFree(buf);
    return equality;
}


extern "C"
int gpu_memcmp(int gpu, void* local, volatile void* remote, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    void* buf = NULL;
    err = hipMalloc(&buf, len);

    if (err != hipSuccess)
    {
        log_debug("Failed to allocate buffer on device, falling back on memcmp");
        return gpu_memcmp_ram(gpu, local, remote, len);
    }

    err = hipMemcpy(buf, (void*) remote, len, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        log_error("Failed to copy from remote buffer to GPU buffer, falling back on memcmp");
        hipFree(buf);
        return gpu_memcmp_ram(gpu, local, remote, len);
    }

    int result = gpu_memcmp_gpu(gpu, local, buf, len);
    hipFree(buf);
    return result;
}


extern "C"
int gpu_device_count()
{
    hipError_t err;
    int count = 0;

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess)
    {
        log_error("Something went wrong: %s", hipGetErrorString(err));
        return -1;
    }

    return count;
}


extern "C"
void* gpu_malloc(int gpu, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }

    void* buf = NULL;
    err = hipMalloc(&buf, len);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate device memory: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("Allocated device buffer %p", buf);
    return buf;
}


extern "C"
void gpu_free(int gpu, void* ptr)
{
    log_debug("Freing buffer %p", ptr);
    hipSetDevice(gpu);
    hipFree(ptr);
}


extern "C"
void devptr_set_sync_memops(void* dev_ptr)
{
    unsigned flag = 1;
    
    hipError_t err = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) dev_ptr);

    if (err != hipSuccess)
    {
        log_error("Failed to set pointer attribute CU_POINTER_ATTRIBYTE_SYNC_MEMOPS");
    }
}


extern "C"
void* gpu_devptr(int gpu, void* ptr)
{
    hipPointerAttribute_t attrs;

    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }
        
    err = hipPointerGetAttributes(&attrs, ptr);
    if (err != hipSuccess)
    {
        log_error("Failed to get pointer attributes: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("CUDA device buffer %p has device ptr %p", ptr, attrs.devicePointer);
    return attrs.devicePointer;
}


extern "C"
void gpu_memcpy_buffer_to_local(int gpu, void* gpu_buf, void* ram_buf, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();

    err = hipMemcpy(ram_buf, gpu_buf, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();
}



extern "C"
void gpu_prepare_memcpy(int gpu)
{
    hipError_t err = hipSetDevice(gpu);

    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
}



extern "C"
void gpu_memcpy_remote_to_local(void* local_buf, volatile void* remote_buf, size_t len)
{
    hipDeviceSynchronize();

    hipError_t err = hipMemcpy(local_buf, (void*) remote_buf, len, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
    }
}


extern "C"
void gpu_memcpy_local_to_remote(void* local_buf, volatile void* remote_buf, size_t len)
{
    hipDeviceSynchronize();

    hipError_t err = hipMemcpy((void*) remote_buf, local_buf, len, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        log_error("%s", hipGetErrorString(err));
    }
}


extern "C"
int gpu_info(int gpu, gpu_info_t* info)
{
    hipError_t err;
    hipDeviceProp_t prop;

    err = hipGetDeviceProperties(&prop, gpu);
    if (err != hipSuccess)
    {
        log_error("Unknown GPU %d: %s", gpu, hipGetErrorString(err));
        return 0;
    }

    info->id = gpu;
    strncpy(info->name, prop.name, 256);
    info->domain = prop.pciBusID;
    info->bus = prop.pciDomainID;
    info->device = prop.pciDeviceID;

    return 1;
}
