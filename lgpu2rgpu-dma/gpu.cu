#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sisci_api.h>
#include <stdint.h>
#include <stdlib.h>
#include "gpu.h"
#include "reporting.h"


__global__ void gpu_memset_kernel(void* buf, size_t len, uint8_t val)
{
    const int num = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int pos = y * (gridDim.x * blockDim.x) + x;

    uint8_t* ptr = (uint8_t*) buf;

    for (size_t i = pos * (len / num), n = (pos + 1) * (len / num); i < n && i < len; ++i)
    {
        ptr[i] = val;
    }

    __syncthreads();
}



extern "C"
void gpu_memset(int gpu, void* ptr, size_t len, uint8_t val)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    dim3 grid;
    grid.x = 4;
    grid.y = 4;

    dim3 block;
    block.x = 4;
    block.y = 4;

    log_debug("Filling buffer with value %02x...", val);

    gpu_memset_kernel<<<grid, block>>>(ptr, len, val);

    hipDeviceSynchronize();
}



extern "C"
size_t gpu_memcmp(int gpu, void* gpuptr, volatile void* ramptr, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    uint8_t* buf = NULL;
    err = hipHostAlloc(&buf, len, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate host memory: %s", hipGetErrorString(err));
        return 0;
    }

    err = hipMemcpy(buf, gpuptr, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return 0;
    }

    hipDeviceSynchronize();

    size_t idx;
    volatile uint8_t* ptr = (volatile uint8_t*) ramptr;

    log_debug("Comparing local GPU memory %p to remote memory %p", gpuptr, ramptr);
    for (idx = 0; idx < len; ++idx)
    {
        if (buf[idx] != ptr[idx])
        {
            log_debug("Byte %lu differs (%02x %02x)", idx, buf[idx], ptr[idx]);
            break;
        }
    }

    hipHostFree(buf);
    return idx;
}



extern "C"
int gpu_device_count()
{
    hipError_t err;
    int count = 0;

    err = hipGetDeviceCount(&count);
    if (err != hipSuccess)
    {
        log_error("Something went wrong: %s", hipGetErrorString(err));
        return -1;
    }

    return count;
}



extern "C"
void* gpu_malloc(int gpu, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }

    void* buf = NULL;
    err = hipMalloc(&buf, len);
    if (err != hipSuccess)
    {
        log_error("Failed to allocate device memory: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("Allocated device buffer %p", buf);
    return buf;
}



extern "C"
void gpu_free(int gpu, void* ptr)
{
    log_debug("Freing buffer %p", ptr);
    hipSetDevice(gpu);
    hipFree(ptr);
}



extern "C"
void devptr_set_sync_memops(void* dev_ptr)
{
    unsigned flag = 1;
    
    hipError_t err = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) dev_ptr);

    if (err != hipSuccess)
    {
        log_error("Failed to set pointer attribute CU_POINTER_ATTRIBYTE_SYNC_MEMOPS");
    }
}



extern "C"
void* gpu_devptr(int gpu, void* ptr)
{
    hipPointerAttribute_t attrs;

    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return NULL;
    }
        
    err = hipPointerGetAttributes(&attrs, ptr);
    if (err != hipSuccess)
    {
        log_error("Failed to get pointer attributes: %s", hipGetErrorString(err));
        return NULL;
    }

    log_debug("CUDA device buffer %p has device ptr %p", ptr, attrs.devicePointer);
    return attrs.devicePointer;
}


extern "C"
void gpu_memcpy_buffer_to_local(int gpu, void* gpu_buf, void* ram_buf, size_t len)
{
    hipError_t err = hipSetDevice(gpu);
    if (err != hipSuccess)
    {
        log_error("Failed to set GPU: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();

    err = hipMemcpy(ram_buf, gpu_buf, len, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        log_error("Failed to memcpy: %s", hipGetErrorString(err));
        return;
    }

    hipDeviceSynchronize();
}
