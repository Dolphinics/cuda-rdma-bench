#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sisci_api.h>
#include "local.h"
#include "log.h"


int main(int argc, char** argv)
{
    sci_error_t err;
    SCIInitialize(0, &err);

    verbosity = 4;

    l_segment_t segment;

    if (CreateLocalSegment(&segment, 10, 0) != 0)
    {
        return 1;
    }

    if (AllocSegmentMem(segment, 4096, 0) != 0)
    {
        RemoveLocalSegment(segment);
        return 1;
    }

    int* ptr = (int*) GetLocalSegmentPtr(segment);
    if (ptr == NULL)
    {
        RemoveLocalSegment(segment);
        return 1;
    }

    *ptr = 0xdeadbeef;

    printf("%x\n", *ptr);
    
    RemoveLocalSegment(segment);
    SCITerminate();

    return 0;
}
